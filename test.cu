#include "hip/hip_runtime.h"
#include<iostream>
#include"set_intersect.cuh"
#include"error.cuh"
#include"time.h"
#include"linear_search.cuh"
#include"gen_test.cuh"

#define MAX_COUNT 100000
#define STEP 10
// #define STEP 10
#define TEST_NUM 3
#define SINGLE_SIZE (MAX_COUNT/STEP)
#define CONTAINER_SIZE (SINGLE_SIZE * TEST_NUM)
// #define TEST_ROUNDS 20
#define TEST_MIN_DEGREE 2
#define TEST_MAX_DEGREE 30

typedef struct{
    double time;
    int ans;
}container;

container cpu_container[CONTAINER_SIZE];

__global__ void test_bs(int *beg_pos_device,int * adj_list_device,int num_vertex,unsigned long long *ans_pos){
    // printf("a1:%d,b1:%d\n",a[1],b[1]);
    // printf("a10:%d,b10:%d\n",a[10],b[10]);
    // printf("sizea:%d,sizeb:%d\n",size_a,size_b);
    __shared__ unsigned long long G_counter[BLOCK_SIZE/WARPSIZE];
    unsigned long long P_counter = 0;
    int warp_id = (threadIdx.x >> 5) & 31;
    int In_warp_id = threadIdx.x & 31;
    if (threadIdx.x == 0)
	{
		ans_pos[0] = 0;
	}
    if(In_warp_id == 0){
        G_counter[warp_id] = 0;
    }
    __syncthreads();
    int now = warp_id;
    while(now < num_vertex){
        int vertex_start = beg_pos_device[now];
        int vertex_degree = beg_pos_device[now + 1] - vertex_start;
        if(vertex_degree >= TEST_MAX_DEGREE || vertex_degree <= TEST_MIN_DEGREE){
            now += BLOCK_SIZE/WARPSIZE;
            continue;
        }
        int warp_iret = vertex_start;
        while(warp_iret < vertex_degree){
            int to_search = adj_list_device[warp_iret];
            int to_search_start = beg_pos_device[to_search];
            int to_search_degree = beg_pos_device[to_search+1] - to_search_start;
            P_counter += intersect_bs(adj_list_device + vertex_start,vertex_degree,adj_list_device + to_search_start, to_search_degree);
            warp_iret += 1;
        }
#ifndef __DYNAMIC
        now += BLOCK_SIZE/WARPSIZE;
#endif
    }
    if(In_warp_id == 31){
        atomicAdd(&G_counter[warp_id],P_counter);
    }
    __syncthreads();
    if (In_warp_id == 0)
	{
		atomicAdd(&ans_pos[0], G_counter[warp_id]);
	}
}

__global__ void test_merge(int *beg_pos_device,int * adj_list_device,int num_vertex,unsigned long long *ans_pos){
    // printf("a1:%d,b1:%d\n",a[1],b[1]);
    // printf("a10:%d,b10:%d\n",a[10],b[10]);
    // printf("sizea:%d,sizeb:%d\n",size_a,size_b);
    __shared__ int G_counter[BLOCK_SIZE/WARPSIZE];
    unsigned long long P_counter = 0;
    int warp_id = (threadIdx.x >> 5) & 31;
    int In_warp_id = threadIdx.x & 31;
    if (threadIdx.x == 0)
	{
		ans_pos[0] = 0;
	}
    if(In_warp_id == 0){
        G_counter[warp_id] = 0;
    }
    __syncthreads();
    int now = warp_id;
    while(now < num_vertex){
        int vertex_start = beg_pos_device[now];
        int vertex_degree = beg_pos_device[now + 1] - vertex_start;
        if(vertex_degree >= TEST_MAX_DEGREE || vertex_degree <= TEST_MIN_DEGREE){
            now += BLOCK_SIZE/WARPSIZE;
            continue;
        }
        int warp_iret = vertex_start;
        while(warp_iret < vertex_degree){
            int to_search = adj_list_device[warp_iret];
            int to_search_start = beg_pos_device[to_search];
            int to_search_degree = beg_pos_device[to_search+1] - to_search_start;
            P_counter += intersect_num_merge(adj_list_device + vertex_start,vertex_degree,adj_list_device + to_search_start, to_search_degree);
            warp_iret += 1;
        }
#ifndef __DYNAMIC
        now += BLOCK_SIZE/WARPSIZE;
#endif
    }
    atomicAdd(&G_counter[warp_id],P_counter);
    __syncthreads();
    if (In_warp_id == 0)
	{
		atomicAdd(&ans_pos[0], G_counter[warp_id]);
	}
}

__global__ void test_linear(int *beg_pos_device,int * adj_list_device,int num_vertex,unsigned long long *ans_pos,int *partition){
    // printf("a1:%d,b1:%d\n",a[1],b[1]);
    // printf("a10:%d,b10:%d\n",a[10],b[10]);
    // printf("sizea:%d,sizeb:%d\n",size_a,size_b);
    __shared__ unsigned long long G_counter[BLOCK_SIZE/WARPSIZE];
    unsigned long long P_counter = 0;
    int warp_id = (threadIdx.x >> 5) & 31;
    int In_warp_id = threadIdx.x & 31;
    if (threadIdx.x == 0)
	{
		ans_pos[0] = 0;
	}
    if(In_warp_id == 0){
        G_counter[warp_id] = 0;
    }
    __syncthreads();
    __shared__ int bin_count[HASH_MAX];
	__shared__ int shared_partition[HASH_MAX * shared_BUCKET_SIZE + 1];//shared hash bin
    int now = warp_id;
    while(now < num_vertex){
        int vertex_start = beg_pos_device[now];
        int vertex_degree = beg_pos_device[now + 1] - vertex_start;
        if(vertex_degree >= TEST_MAX_DEGREE || vertex_degree <= TEST_MIN_DEGREE){
            now += BLOCK_SIZE/WARPSIZE;
            continue;
        }
        int warp_iret = vertex_start + warp_id;
        while(warp_iret < vertex_degree){
            int to_search = adj_list_device[warp_iret];
            int to_search_start = beg_pos_device[to_search];
            int to_search_degree = beg_pos_device[to_search+1] - to_search_start;
            P_counter += single_search_warp_static(shared_partition,partition,bin_count,adj_list_device + vertex_start,vertex_degree,adj_list_device + to_search_start, to_search_degree);
            warp_iret += 1;
        }
#ifndef __DYNAMIC
        now += BLOCK_SIZE/WARPSIZE;
#endif
    }
    atomicAdd(&G_counter[warp_id],P_counter);
    __syncthreads();
    if (In_warp_id == 0)
	{
		atomicAdd(&ans_pos[0], G_counter[warp_id]);
	}
}

void test(int *beg_pos,int *adj_list,int num_vertex,int num_edge){
    
    int *beg_pos_device;
    int *adj_list_device;
    unsigned long long *ans_pos;
    unsigned long long  cpu_ans;

    HRR(hipMalloc((void **)&beg_pos_device,sizeof(int)*(num_vertex + 1)));
    HRR(hipMalloc((void **)&adj_list_device,sizeof(int)*num_edge));
    HRR(hipMalloc((void **)&ans_pos,sizeof(unsigned long long )));

    HRR(hipMemcpy(beg_pos_device, beg_pos, sizeof(int)*(num_vertex + 1), hipMemcpyHostToDevice));
    HRR(hipMemcpy(adj_list_device, adj_list, sizeof(int)*num_edge, hipMemcpyHostToDevice));

    //bs test
    double time_start = clock();
    test_bs<<<1, BLOCK_SIZE>>>(beg_pos_device, adj_list_device, num_vertex,ans_pos);
    HRR(hipDeviceSynchronize());
    double cmp_time = clock() - time_start;
    double cmptime = cmp_time / CLOCKS_PER_SEC;
    cpu_container[0].time = cmptime;
    HRR(hipMemcpy(&cpu_ans, ans_pos , sizeof(unsigned long long ), hipMemcpyDeviceToHost));
    cpu_container[0].ans = cpu_ans;

    //merge test
    time_start = clock();
    test_merge<<<1, BLOCK_SIZE>>>(beg_pos_device, adj_list_device, num_vertex,ans_pos);
    HRR(hipDeviceSynchronize());
    cmp_time = clock() - time_start;
    cmptime = cmp_time / CLOCKS_PER_SEC;
    cpu_container[SINGLE_SIZE].time = cmptime;
    HRR(hipMemcpy(&cpu_ans, ans_pos , sizeof(unsigned long long ), hipMemcpyDeviceToHost));
    cpu_container[SINGLE_SIZE].ans = cpu_ans;

    int *partition_gpu;
    HRR(hipMalloc((void **)&partition_gpu,sizeof(int)*1024*HASH_MAX));

    time_start = clock();
    dynamic_assign<<<1, BLOCK_SIZE>>>(adj_list_device, beg_pos_device,num_edge,num_vertex,partition_gpu, ans_pos);
    HRR(hipDeviceSynchronize());
    cmp_time = clock() - time_start;
    cmptime = cmp_time / CLOCKS_PER_SEC;
    cpu_container[SINGLE_SIZE * 2].time = cmptime;
    HRR(hipMemcpy(&cpu_ans, ans_pos , sizeof(unsigned long long ), hipMemcpyDeviceToHost));
    cpu_container[SINGLE_SIZE * 2].ans = cpu_ans;
    
    HRR(hipFree(beg_pos_device));
    HRR(hipFree(adj_list_device));
    HRR(hipFree(ans_pos));
    HRR(hipFree(partition_gpu));
}

void write_results_to_file(char *filename){
    FILE *file = fopen(filename, "w"); // 打开文件以便写入
    if(file == NULL){
        printf("Error opening file!\n");
        return;
    }
    fprintf(file, "gold_ans\tsize\tbs\tTime(s)\tCorrectness\tmerge\tTime(s)\tCorrectness\tlinear\tTime(s)\tCorrectness\n"); // 写入表头
    for(int i = 0; i < SINGLE_SIZE ; i++){
        fprintf(file, "%10d\t%10f\t%1d\t\t%10f\t%1d\t\t%10f\t%1d\n", 
        i*STEP,cpu_container[i].time, cpu_container[i].ans,cpu_container[SINGLE_SIZE+i].time,cpu_container[SINGLE_SIZE+i].ans,cpu_container[2*SINGLE_SIZE+i].time,cpu_container[2*SINGLE_SIZE+i].ans); // 将每个元素写入文件
    }
    fclose(file); // 关闭文件
}

int main(int argc,char ** argv ){
    // generate_test(a,MAX_COUNT);
    // generate_test(b,MAX_COUNT);
    int *beg_pos;//list of the start of the vertex
    int *adj_list;//list of the end of the edge
    int num_edge;
    int num_vertex;
    gen_test(argv[1],&beg_pos,&adj_list,&num_vertex,&num_edge);
    
    test(beg_pos,adj_list,num_vertex,num_edge);
    write_results_to_file(argv[2]);
}