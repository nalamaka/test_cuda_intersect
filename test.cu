#include "hip/hip_runtime.h"
#include<iostream>
#include"set_intersect.cuh"
#include"error.cuh"
#include"time.h"
#include"linear_search.cuh"
#include"gen_test.cuh"

#define MAX_COUNT 100000
#define STEP 10
// #define STEP 10
#define TEST_NUM 3
#define SINGLE_SIZE (MAX_COUNT/STEP)
#define CONTAINER_SIZE (SINGLE_SIZE * TEST_NUM)
// #define TEST_ROUNDS 20

typedef struct{
    double time;
    int ans;
}container;

int a[MAX_COUNT];
int b[MAX_COUNT];
int gold_ans[SINGLE_SIZE];
container cpu_container[CONTAINER_SIZE];

void generate_test(int *a,int count){
    for(int i=0;i<count;i++){
        a[i] = i;
    }
}

void gen_gold_once(int *a,int size_a,int *b,int size_b,int *ans,int pos){
    int temp = 0;
    for(int i=0;i<size_a;i++){
        for(int j=0;j<size_b;j++){
            if(a[i] == b[j]){
                temp = temp + 1;
                break;
            }
        }
    }
    ans[pos] = temp;
}

void generate_gold_ans(int *a,int size_a,int *b,int size_b,int gold_step,int *gold_container){
    // int end = size_a / gold_step;
    for(int curr_pos = 0;curr_pos < size_a;curr_pos += gold_step){
        if(curr_pos > size_b){
            gen_gold_once(a,curr_pos,b,size_b,gold_container,curr_pos/gold_step);
        }else{
            gen_gold_once(a,curr_pos,b,curr_pos,gold_container,curr_pos/gold_step);
        }
    }
}
__global__ void test_bs(int *a,int size_a,int *b,int size_b,int *ans_pos){
    // printf("a1:%d,b1:%d\n",a[1],b[1]);
    // printf("a10:%d,b10:%d\n",a[10],b[10]);
    // printf("sizea:%d,sizeb:%d\n",size_a,size_b);
    __shared__ int G_counter;
    int P_counter;
    if (threadIdx.x == 0)
	{
		ans_pos[0] = 0;
        G_counter = 0;
	}
    __syncthreads();
#ifndef TEST_ROUNDS
    P_counter = intersect_bs_cache(a,size_a,b,size_b);
#else
    for(int i=0;i < TEST_ROUNDS;i++){
        P_counter = intersect_bs_cache(a,size_a,b,size_b);
    }
#endif
    // printf("Pcounter:%d\n",P_counter);
    atomicMax(&G_counter,P_counter);
    __syncwarp();
    if (threadIdx.x  == 0)
	{
		atomicAdd(&ans_pos[0], G_counter);
	}
}

__global__ void test_merge(int *a,int size_a,int *b,int size_b,int *ans_pos){
    // printf("a1:%d,b1:%d\n",a[1],b[1]);
    // printf("a10:%d,b10:%d\n",a[10],b[10]);
    // printf("sizea:%d,sizeb:%d\n",size_a,size_b);
    __shared__ int G_counter;
    int P_counter;
    if (threadIdx.x == 0)
	{
		ans_pos[0] = 0;
        G_counter = 0;
	}
    __syncthreads();
#ifndef TEST_ROUNDS
    P_counter = intersect_num_merge(a,size_a,b,size_b);
    // printf("Pcounter:%d\n",P_counter);
#else
    for(int i=0;i < TEST_ROUNDS;i++){
        P_counter = intersect_num_merge(a,size_a,b,size_b);
    }
#endif
    atomicAdd(&G_counter,P_counter);
    __syncwarp();
    if (threadIdx.x == 0)
	{
		atomicAdd(&ans_pos[0], G_counter);
	}
}

__global__ void test_linear(int *a,int size_a,int *b,int size_b,int *ans_pos,int *partition){
    // printf("a1:%d,b1:%d\n",a[1],b[1]);
    // printf("a10:%d,b10:%d\n",a[10],b[10]);
    // printf("sizea:%d,sizeb:%d\n",size_a,size_b);
    __shared__ int G_counter;
    int P_counter;
    if (threadIdx.x == 0)
	{
		ans_pos[0] = 0;
        G_counter = 0;
	}
    __syncthreads();
    __shared__ int bin_count[HASH_MAX];
	__shared__ int shared_partition[HASH_MAX * shared_BUCKET_SIZE + 1];
#ifndef TEST_ROUNDS
    P_counter = single_search_warp_static(shared_partition,partition,bin_count,a,size_a,b,size_b);
#else
    // __shared__ int bin_count[HASH_MAX];
	// __shared__ int shared_partition[HASH_MAX * shared_BUCKET_SIZE + 1];
	// gen_bin(a, size_a, shared_partition,partition,bin_count);
	// __syncwarp();
    // for(int i=0;i < TEST_ROUNDS;i++){
    //     P_counter = single_search_static(shared_partition,partition,bin_count,b,size_b);
    // }
#endif
    atomicAdd(&G_counter,P_counter);
    __syncwarp();
    if (threadIdx.x == 0)
	{
		atomicAdd(&ans_pos[0], G_counter);
	}
}

void test(int count_a,int count_b){
    
    int *a_device;
    int *b_device;
    int *ans_pos;
    int cpu_ans;

    HRR(hipMalloc((void **)&a_device,sizeof(int)*MAX_COUNT));
    HRR(hipMalloc((void **)&b_device,sizeof(int)*MAX_COUNT));
    HRR(hipMalloc((void **)&ans_pos,sizeof(int)*MAX_COUNT));

    HRR(hipMemcpy(a_device, a, sizeof(int) * MAX_COUNT, hipMemcpyHostToDevice));
    HRR(hipMemcpy(b_device, b, sizeof(int) * MAX_COUNT, hipMemcpyHostToDevice));

    for(int test_size = 0; test_size < count_a && test_size < count_b;test_size += STEP){
        double time_start = clock();
        test_bs<<<1, BLOCK_SIZE>>>(a_device, test_size, b_device, test_size,ans_pos);
        HRR(hipDeviceSynchronize());
        double cmp_time = clock() - time_start;
        double cmptime = cmp_time / CLOCKS_PER_SEC;
        cpu_container[test_size/STEP].time = cmptime;
        HRR(hipMemcpy(&cpu_ans, ans_pos , sizeof(int), hipMemcpyDeviceToHost));
        cpu_container[test_size/STEP].ans = cpu_ans;

    }

    for(int test_size = 0; test_size < count_a && test_size < count_b;test_size += STEP){
        double time_start = clock();
        test_merge<<<1, BLOCK_SIZE>>>(a_device, test_size, b_device, test_size,ans_pos);
        HRR(hipDeviceSynchronize());
        double cmp_time = clock() - time_start;
        double cmptime = cmp_time / CLOCKS_PER_SEC;
        cpu_container[SINGLE_SIZE + test_size/STEP].time = cmptime;
        HRR(hipMemcpy(&cpu_ans, ans_pos , sizeof(int), hipMemcpyDeviceToHost));
        cpu_container[SINGLE_SIZE+test_size/STEP].ans = cpu_ans;
    }

    int *partition_gpu;
    HRR(hipMalloc((void **)&partition_gpu,sizeof(int)*1024*HASH_MAX));

    for(int test_size = 0; test_size < count_a && test_size < count_b;test_size += STEP){
        double time_start = clock();
        test_linear<<<1, BLOCK_SIZE>>>(a_device, test_size, b_device, test_size,ans_pos,partition_gpu);
        HRR(hipDeviceSynchronize());
        double cmp_time = clock() - time_start;
        double cmptime = cmp_time / CLOCKS_PER_SEC;
        cpu_container[SINGLE_SIZE * 2 + test_size/STEP].time = cmptime;
        HRR(hipMemcpy(&cpu_ans, ans_pos , sizeof(int), hipMemcpyDeviceToHost));
        cpu_container[SINGLE_SIZE * 2+test_size/STEP].ans = cpu_ans;
    }
    
    HRR(hipFree(a_device));
    HRR(hipFree(b_device));
}

void write_results_to_file(char *filename){
    FILE *file = fopen(filename, "w"); // 打开文件以便写入
    if(file == NULL){
        printf("Error opening file!\n");
        return;
    }
    fprintf(file, "gold_ans\tsize\tbs\tTime(s)\tCorrectness\tmerge\tTime(s)\tCorrectness\tlinear\tTime(s)\tCorrectness\n"); // 写入表头
    for(int i = 0; i < SINGLE_SIZE ; i++){
        fprintf(file, "%10d\t%10d\t%10f\t%1d\t\t%10f\t%1d\t\t%10f\t%1d\n", 
        gold_ans[i],i*STEP,cpu_container[i].time, cpu_container[i].ans,cpu_container[SINGLE_SIZE+i].time,cpu_container[SINGLE_SIZE+i].ans,cpu_container[2*SINGLE_SIZE+i].time,cpu_container[2*SINGLE_SIZE+i].ans); // 将每个元素写入文件
    }
    fclose(file); // 关闭文件
}

int main(int argc,char ** argv ){
    // generate_test(a,MAX_COUNT);
    // generate_test(b,MAX_COUNT);
    int count_a = 100;
    int count_b;
    gen_test(argv[1],a,&count_a,b,&count_b);
    // generate_test(a,count_a);
    // generate_test(b,count_b);
    generate_gold_ans(a,count_a,b,count_b,STEP,gold_ans);
    
    test(count_a,count_b);

    write_results_to_file(argv[2]);
}