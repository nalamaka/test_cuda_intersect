#include "hip/hip_runtime.h"
#include<iostream>
#include"set_intersect.cuh"
#include"error.cuh"
#include"time.h"

#define MAX_COUNT 100000
#define STEP 1000
#define TEST_NUM 3
#define SINGLE_SIZE (MAX_COUNT/STEP)
#define CONTAINER_SIZE (SINGLE_SIZE * TEST_NUM)

typedef struct{
    double time;
    int TorF;
}container;

int a[MAX_COUNT];
int b[MAX_COUNT];
int gold_ans[SINGLE_SIZE];
container cpu_container[CONTAINER_SIZE];

void generate_test(int *a,int count){
    for(int i=0;i<count;i++){
        a[i] = i;
    }
}

void gen_gold_once(int *a,int size_a,int *b,int size_b,int *ans,int pos){
    ans[pos] = pos * STEP;
}

void generate_gold_ans(int *a,int size_a,int *b,int size_b,int gold_step,int *gold_container){
    int end = size_a / gold_step;
    for(int curr_pos = 0;curr_pos < size_a;curr_pos += gold_step){
        gen_gold_once(a,curr_pos,b,curr_pos,gold_container,curr_pos/gold_step);
    }
}

void write_results_to_file(int *results, int size, const char *filename){
    FILE *file = fopen(filename, "w"); // 打开文件以便写入
    if(file == NULL){
        printf("Error opening file!\n");
        return;
    }
    for(int i = 0; i < size; i++){
        fprintf(file, "%d\n", results[i]); // 将每个元素写入文件
    }
    fclose(file); // 关闭文件
}

__global__ void test_bs(int *a,int size_a,int *b,int size_b,int *ans_pos){
    printf("a1:%d,b1:%d\n",a[1],b[1]);
    printf("a10:%d,b10:%d\n",a[10],b[10]);
    printf("sizea:%d,sizeb:%d\n",size_a,size_b);
    __shared__ int G_counter;
    if (threadIdx.x == 0)
	{
		ans_pos[0] = 0;
        G_counter = 0;
	}
    __syncwarp();
    int P_counter = intersect_bs_cache(a,size_a,b,size_b);
    printf("Pcounter:%d\n",P_counter);
    atomicAdd(&G_counter,P_counter);
    __syncwarp();
    if (threadIdx.x == 0)
	{
		atomicAdd(&ans_pos[0], G_counter);
	}
}

void test(){
    
    int *a_device;
    int *b_device;
    int *ans_pos;
    int cpu_ans;

    HRR(hipMalloc((void **)&a_device,sizeof(int)*MAX_COUNT));
    HRR(hipMalloc((void **)&b_device,sizeof(int)*MAX_COUNT));
    HRR(hipMalloc((void **)&ans_pos,sizeof(int)*MAX_COUNT));

    HRR(hipMemcpy(a_device, a, sizeof(int) * MAX_COUNT, hipMemcpyHostToDevice));
    HRR(hipMemcpy(b_device, b, sizeof(int) * MAX_COUNT, hipMemcpyHostToDevice));

    for(int test_size = 0; test_size < STEP * 2;test_size += STEP){
        double time_start = clock();
        test_bs<<<1, 32>>>(a_device, test_size, b_device, test_size,ans_pos);
        HRR(hipDeviceSynchronize());
        double cmp_time = clock() - time_start;
        double cmptime = cmp_time / CLOCKS_PER_SEC;
        cpu_container[test_size/STEP].time = cmptime;
        HRR(hipMemcpy(&cpu_ans, ans_pos , sizeof(int), hipMemcpyDeviceToHost));
        if(cpu_ans == gold_ans[test_size/STEP]){
            cpu_container[test_size/STEP].TorF = 1;
        }else{
            printf("wrong ans is %d\n",cpu_ans);
            cpu_container[test_size/STEP].TorF = 0;
        }

    }
    
    HRR(hipFree(a_device));
    HRR(hipFree(b_device));
}

void write_results_to_file(){
    FILE *file = fopen("./out.txt", "w"); // 打开文件以便写入
    if(file == NULL){
        printf("Error opening file!\n");
        return;
    }
    fprintf(file, "Time(s),Correctness\n"); // 写入表头
    for(int i = 0; i < CONTAINER_SIZE ; i++){
        fprintf(file, "%f,%d\n", cpu_container[i].time, cpu_container[i].TorF); // 将每个元素写入文件
    }
    fclose(file); // 关闭文件
}

int main(){
    generate_test(a,MAX_COUNT);
    generate_test(b,MAX_COUNT);
    generate_gold_ans(a,MAX_COUNT,b,MAX_COUNT,STEP,gold_ans);
    
    test();

    write_results_to_file();
}