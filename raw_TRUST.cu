
#include <hip/hip_runtime.h>
#include<iostream>
#define shared_BUCKET_SIZE 6
#define SUM_SIZE 1
#define USE_CTA 100
#define USE_WARP 2
#define without_combination 1
#define MIN_DEGREE 2
#define MAX_DEGREE 30
#define HASE_BIT_SIZE 10
#define HASH_MAX (1 << HASE_BIT_SIZE)
#define warp_bucketnum 32
__device__ int linear_search(int neighbor, int *shared_partition, int *partition, int *bin_count, int bin, int BIN_START){

	int i = bin;
	int len = bin_count[i];
	// unsigned int guess = guess_bin[bin];
	// unsigned int neighbor_guess = 1 << ((neighbor >> HASE_BIT_SIZE) & (31));
	// if((guess & neighbor_guess) == 0){
	// 	return 0;
	// }
	int step = 0;
	int nowlen;
	if (len < shared_BUCKET_SIZE)//maximum len to search in the shared bin
		nowlen = len;
	else
		nowlen = shared_BUCKET_SIZE;
	while (step < nowlen)
	{
		if (shared_partition[i] == neighbor)
		{
			return 1;
		}
		i += HASH_MAX;
		step += 1;
	}

	len -= shared_BUCKET_SIZE;
	i = bin + BIN_START;
	step = 0;
	while (step < len)
	{
		if (partition[i] == neighbor)
		{
			return 1;
		}
		i += HASH_MAX;
		step += 1;
	}
	return 0;
	//if a vertex cannot find in the hash bin it may cost too much time to find the fact
	//also it not idicate a node that is not valid
}
__global__ void dynamic_assign(int *adj_list, int *beg_pos, int edge_count, int vertex_count, int *partition, unsigned long long *GLOBAL_COUNT)
{

	// int tid=threadIdx.x+blockIdx.x*blockDim.x;
	__shared__ int bin_count[HASH_MAX];
	__shared__ int shared_partition[HASH_MAX * shared_BUCKET_SIZE + 1];
	
	// __shared__ int shared_now,shared_workid;
	// __shared__ int useless[1024*9];
	// useless[threadIdx.x]=1;
	unsigned long long __shared__ G_counter;
	int WARPSIZE = 32;
	if (threadIdx.x == 0)
	{
		G_counter = 0;
	}
	// timetest
	unsigned long long TT = 0, HT = 0, IT = 0;
	unsigned long long __shared__ G_TT, G_HT, G_IT;
	G_TT = 0, G_HT = 0, G_IT = 0;

	int BIN_START = blockIdx.x * HASH_MAX * 100;
	// __syncthreads();
	unsigned long long P_counter = 0;

	// unsigned long long start_time;

	// start_time = clock64();
	// CTA for large degree vertex
	int vertex = (blockIdx.x * 1 + 0) * 1;//the first vertex to search
	int vertex_end = vertex + 1;
	__shared__ int ver;
	while (vertex < 0)
	{
		int degree = beg_pos[vertex + 1] - beg_pos[vertex];//the degree of a node to search
		int start = beg_pos[vertex];
		int end = beg_pos[vertex + 1];
		int now = threadIdx.x + start;//divide the task to the whole warp
		int MODULO = HASH_MAX - 1;
		int BIN_OFFSET = 0;
		// clean bin_count
		for (int i = threadIdx.x; i < HASH_MAX; i += blockDim.x){
			bin_count[i] = 0;
			// bin_guess[i] = 0;
		}
		__syncthreads();

		// count hash bin
		while (now < end)
		{
			int temp = adj_list[now];
			int bin = temp & MODULO;//hash the temps
			int index;
			// index = atomicAdd(&bin_count[bin], 1);//bin++
			// index = ++bin_count[bin];
			// atomicOr(&bin_guess[bin],or_bits);
			// atomicAdd(&bin_guess[bin],1);
			if (index < shared_BUCKET_SIZE)//index can fit in the shared bucket
			{
				shared_partition[index * HASH_MAX + bin] = temp;
			}
			else if (index < 100)
			{
				index = index - shared_BUCKET_SIZE;
				partition[index * HASH_MAX + bin + BIN_START] = temp;
			}
			now += blockDim.x;
		}
		__syncthreads();
		
		now = threadIdx.x + start;//divide the task to the whole warp
		// while(now < end){
		// 	int temp = adj_list[now];
		// 	int bin = temp & MODULO;
		// 	unsigned int or_bits = 1 << ((temp >> HASE_BIT_SIZE) & (31));
		// 	int ans = atomicOr(&bin_guess[bin],or_bits);
		// 	now += blockDim.x;
		// }

		// unsigned long long hash_time=clock64()-start_time;
		// start_time = clock64();
		// list intersection
		now = beg_pos[vertex];
		end = beg_pos[vertex + 1];
		if (without_combination)
		{
			while (now < end)
			{
				int neighbor = adj_list[now];
				int neighbor_start = beg_pos[neighbor];
				int neighbor_end = beg_pos[neighbor + 1];
				int neighbor_now = neighbor_start + threadIdx.x;
				while (neighbor_now < neighbor_end)
				{
					int temp = adj_list[neighbor_now];
					int bin = temp & MODULO;
					P_counter += linear_search(temp, shared_partition, partition, bin_count, bin + BIN_OFFSET, BIN_START);
					neighbor_now += blockDim.x;
				}
				now++;
			}
		}
		else
		{
			int superwarp_ID = threadIdx.x / 64;
			int superwarp_TID = threadIdx.x % 64;
			int workid = superwarp_TID;
			now = now + superwarp_ID;
			int neighbor = adj_list[now];
			int neighbor_start = beg_pos[neighbor];
			int neighbor_degree = beg_pos[neighbor + 1] - neighbor_start;
			while (now < end)
			{
				while (now < end && workid >= neighbor_degree)
				{
					now += 16;
					workid -= neighbor_degree;
					neighbor = adj_list[now];
					neighbor_start = beg_pos[neighbor];
					neighbor_degree = beg_pos[neighbor + 1] - neighbor_start;
				}
				if (now < end)
				{
					int temp = adj_list[neighbor_start + workid];
					int bin = temp & MODULO;
					P_counter += linear_search(temp, shared_partition, partition, bin_count, bin + BIN_OFFSET, BIN_START);
				}
				workid += 64;
			}
		}
		__syncthreads();
		vertex += gridDim.x * 1;
	}
	// warp method
	int WARPID = threadIdx.x / WARPSIZE;
	int WARP_TID = threadIdx.x % WARPSIZE;
	int WARPDIM = blockDim.x * gridDim.x / WARPSIZE;
	vertex =WARPID + blockIdx.x * blockDim.x / WARPSIZE;
	vertex_end = vertex + 1;
	while (vertex < vertex_count){
		// unsigned long long start_time = clock64();
		int degree = beg_pos[vertex + 1] - beg_pos[vertex];
		if (degree < MIN_DEGREE || degree > MAX_DEGREE){
			vertex += WARPDIM * 1;
			continue;
		}
		int start = beg_pos[vertex];
		int end = beg_pos[vertex + 1];
		int now = WARP_TID + start;
		int MODULO = warp_bucketnum - 1;
		int BIN_OFFSET = WARPID * warp_bucketnum;

		for (int i = BIN_OFFSET + WARP_TID; i < BIN_OFFSET + warp_bucketnum; i += WARPSIZE){
			bin_count[i] = 0;
		}
		__syncwarp();

		// count hash bin
		while (now < end)
		{
			int temp = adj_list[now];
			int bin = temp & MODULO;
			bin += BIN_OFFSET;
			int index;
			index = atomicAdd(&bin_count[bin], 1);
			// index = ++bin_count[bin];
			// atomicAdd(&bin_guess[bin],1);
			// atomicOr(&bin_guess[bin],or_bits);
			if (index < shared_BUCKET_SIZE)
			{
				shared_partition[index * HASH_MAX + bin] = temp;
			}
			else if (index < 100)
			{
				index = index - shared_BUCKET_SIZE;
				partition[index * HASH_MAX + bin + BIN_START] = temp;
			}
			now += WARPSIZE;
		}
		__syncwarp();
		now = threadIdx.x + start;//divide the task to the whole warp
		// while(now < end){
		// 	int temp = adj_list[now];
		// 	int bin = temp & MODULO;
		// 	unsigned int or_bits = 1 << ((temp >> HASE_BIT_SIZE) & (31));
		// 	int ans = atomicOr(&bin_guess[bin],or_bits);
		// 	now += blockDim.x;
		// }

		// list intersection
		now = beg_pos[vertex];
		end = beg_pos[vertex + 1];

		if (without_combination)
		{
			while (now < end)
			{
				int neighbor = adj_list[now];
				int neighbor_start = beg_pos[neighbor];
				int neighbor_end = beg_pos[neighbor + 1];
				int neighbor_now = neighbor_start + WARP_TID;
				while (neighbor_now < neighbor_end)
				{
					int temp = adj_list[neighbor_now];
					int bin = temp & MODULO;
					P_counter += linear_search(temp, shared_partition, partition, bin_count, bin + BIN_OFFSET, BIN_START);
					neighbor_now += WARPSIZE;
				}
				now++;
			}
		}
		else
		{
			int workid = WARP_TID;
			while (now < end)
			{
				int neighbor = adj_list[now];
				int neighbor_start = beg_pos[neighbor];
				int neighbor_degree = beg_pos[neighbor + 1] - neighbor_start;

				while (now < end && workid >= neighbor_degree)
				{
					now++;
					workid -= neighbor_degree;
					neighbor = adj_list[now];
					neighbor_start = beg_pos[neighbor];
					neighbor_degree = beg_pos[neighbor + 1] - neighbor_start;
				}
				if (now < end)
				{
					int temp = adj_list[neighbor_start + workid];
					int bin = temp & MODULO;
					P_counter += linear_search(temp, shared_partition, partition, bin_count, bin + BIN_OFFSET, BIN_START);
				}
				__syncwarp();
				now = __shfl_sync(0xffffffff, now, 31);
				workid = __shfl_sync(0xffffffff, workid, 31);

				workid += WARP_TID + 1;

				// workid+=WARPSIZE;
			}
		}
		__syncwarp();
		vertex += WARPDIM;
	}
	atomicAdd(&G_counter, P_counter);

	__syncthreads();
	if (threadIdx.x == 0)
	{
		atomicAdd(&GLOBAL_COUNT[0], G_counter);
	}
}